#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 32
#define OUTPUT_SIZE 10
#define TRAIN_SIZE 10000
#define TEST_SIZE 1000
#define BATCH_SIZE 4
#define EPOCHS 3
#define LEARNING_RATE 0.001

typedef struct {
    float *weights1;
    float *weights2;
    float *bias1;
    float *bias2;
    float *grad_weights1;
    float *grad_weights2;
    float *grad_bias1;
    float *grad_bias2;
} NeuralNetwork;

// Modify the CUDA_CHECK macro to print more information
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            hipDeviceReset(); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// load batched img data
void load_data(const char *filename, float *data, int size) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(1);
    }
    size_t read_size = fread(data, sizeof(float), size, file);
    if (read_size != size) {
        fprintf(stderr, "Error reading data: expected %d elements, got %zu\n", size, read_size);
        exit(1);
    }
    fclose(file);
}

// load batch labels
void load_labels(const char *filename, int *labels, int size) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(1);
    }
    size_t read_size = fread(labels, sizeof(int), size, file);
    if (read_size != size) {
        fprintf(stderr, "Error reading labels: expected %d elements, got %zu\n", size, read_size);
        exit(1);
    }
    fclose(file);
}

// kaiming init func for weights
void initialize_weights(float *weights, int size) {
    float scale = sqrtf(2.0f / size);
    for (int i = 0; i < size; i++) {
        weights[i] = ((float)rand() / RAND_MAX) * scale - (scale / 2.0f);
    }
}

// basic init for biases
void initialize_bias(float *bias, int size) {
    for (int i = 0; i < size; i++) {
        bias[i] = 0.0f;
    }
}

// CUDA kernel for matrix multiplication (A @ B)
__global__ void matmul_a_b_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// CUDA kernel for matrix multiplication (A @ B.T)
__global__ void matmul_a_bt_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[col * n + i];
        }
        C[row * k + col] = sum;
    }
}

// CUDA kernel for matrix multiplication (A.T @ B)
__global__ void matmul_at_b_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < m; ++i) {
            sum += A[i * n + row] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// CUDA kernel for ReLU activation
__global__ void relu_kernel(float *x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] = fmaxf(0.0f, x[idx]);
    }
}

// CUDA kernel for bias addition
__global__ void bias_add_kernel(float *x, float *bias, int batch_size, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int b = idx / size;
    int i = idx % size;

    if (b < batch_size && i < size) {
        x[idx] += bias[i];
    }
}

// CUDA kernel for softmax
__global__ void softmax_kernel(float *x, int batch_size, int size) {
    int b = blockIdx.x;
    if (b < batch_size) {
        float max_val = x[b * size];
        for (int i = 1; i < size; ++i) {
            max_val = fmaxf(max_val, x[b * size + i]);
        }

        float sum = 0.0f;
        for (int i = 0; i < size; ++i) {
            x[b * size + i] = expf(x[b * size + i] - max_val);
            sum += x[b * size + i];
        }

        for (int i = 0; i < size; ++i) {
            x[b * size + i] = fmaxf(x[b * size + i] / sum, 1e-7f);
        }
    }
}

__global__ void clip_gradients_kernel(float *gradients, int size, float max_norm) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float grad = gradients[idx];
        if (grad > max_norm) {
            gradients[idx] = max_norm;
        } else if (grad < -max_norm) {
            gradients[idx] = -max_norm;
        }
    }
}


// Modified forward function using CUDA kernels
void forward(NeuralNetwork *nn, float *d_input, float *d_hidden, float *d_output, int batch_size) {
    // 1024 threads/blocks
    dim3 block_size(32, 32);
    // just enough blocks + threads for our naive matmul kernel
    dim3 grid_size((HIDDEN_SIZE + block_size.x - 1) / block_size.x, (batch_size + block_size.y - 1) / block_size.y);

    // Input to Hidden (X @ W1)
    matmul_a_b_kernel<<<grid_size, block_size>>>(d_input, nn->weights1, d_hidden, batch_size, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Add bias1 (one bias term for each neuron (multiple weights))
    bias_add_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, nn->bias1, batch_size, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Apply ReLU
    relu_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, batch_size * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Hidden to Output (Hidden @ W2)
    grid_size.x = (OUTPUT_SIZE + block_size.x - 1) / block_size.x;
    grid_size.y = (batch_size + block_size.y - 1) / block_size.y;
    matmul_a_b_kernel<<<grid_size, block_size>>>(d_hidden, nn->weights2, d_output, batch_size, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Add bias2 (also one bias term per neuron)
    bias_add_kernel<<<(batch_size * OUTPUT_SIZE + 255) / 256, 256>>>(d_output, nn->bias2, batch_size, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Apply softmax
    softmax_kernel<<<batch_size, 1>>>(d_output, batch_size, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());
}

// Modify cross_entropy_loss to work with batches (w/out softmax because we already do this in the forward pass)
float cross_entropy_loss(float *output, int *labels, int batch_size) {
    float total_loss = 0.0f;
    for (int b = 0; b < batch_size; b++) {
        total_loss -= logf(fmaxf(output[b * OUTPUT_SIZE + labels[b]], 1e-7f));
    }
    return total_loss / batch_size;
}

// Add this CUDA kernel to zero out gradients
__global__ void zero_grad_kernel(float *grad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad[idx] = 0.0f;
    }
}

// CUDA kernel for computing output gradients
__global__ void compute_output_gradients_kernel(float *grad_output, float *output, int *labels, int batch_size) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch_size) {
        for (int i = 0; i < OUTPUT_SIZE; ++i) {
            grad_output[b * OUTPUT_SIZE + i] = output[b * OUTPUT_SIZE + i];
        }
        grad_output[b * OUTPUT_SIZE + labels[b]] -= 1.0f;
    }
}

// CUDA kernel for updating gradients
__global__ void update_gradients_kernel(float *grad_weights, float *grad_bias, float *grad_layer, float *prev_layer, int batch_size, int prev_size, int curr_size) {
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < curr_size && j < prev_size) {
        float grad_w_sum = 0.0f;
        for (int b = 0; b < batch_size; ++b) {
            grad_w_sum += grad_layer[b * curr_size + i] * prev_layer[b * prev_size + j];
        }
        atomicAdd(&grad_weights[i * prev_size + j], grad_w_sum);

        if (j == 0) {
            float grad_b_sum = 0.0f;
            for (int b = 0; b < batch_size; ++b) {
                grad_b_sum += grad_layer[b * curr_size + i];
            }
            atomicAdd(&grad_bias[i], grad_b_sum);
        }
    }
}

__global__ void drelu_kernel(float *x, float *d_ReLU_out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_ReLU_out[idx] = x[idx] > 0.0f ? 1.0f : 0.0f;
    }
}

// Element-wise multiplication of d_dX2 and d_grad_hidden
__global__ void multiply_gradients_kernel(float *grad1, float *grad2, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad1[idx] *= grad2[idx];
    }
}

// Modified backward function using CUDA kernels
// shape rotating is on par with the visual example (excalidraw diagram) in the mnist-cuda git repo (also found in "assets")
void backward(NeuralNetwork *nn, float *d_input, float *d_hidden, float *d_output, int *d_labels, int batch_size) {
    // Initialize gradients to zero using CUDA kernel

    zero_grad_kernel<<<(HIDDEN_SIZE * INPUT_SIZE + 256 - 1) / 256, 256>>>(nn->grad_weights1, HIDDEN_SIZE * INPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    zero_grad_kernel<<<(OUTPUT_SIZE * HIDDEN_SIZE + 256 - 1) / 256, 256>>>(nn->grad_weights2, OUTPUT_SIZE * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    zero_grad_kernel<<<(HIDDEN_SIZE + 256 - 1) / 256, 256>>>(nn->grad_bias1, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    zero_grad_kernel<<<(OUTPUT_SIZE + 256 - 1) / 256, 256>>>(nn->grad_bias2, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());
    
    // Compute gradients for output layer
    float *d_grad_output;
    CUDA_CHECK(hipMalloc(&d_grad_output, batch_size * OUTPUT_SIZE * sizeof(float)));
    compute_output_gradients_kernel<<<(batch_size + 255) / 256, 256>>>(d_grad_output, d_output, d_labels, batch_size);
    CUDA_CHECK(hipGetLastError());

    // Update gradients for weights2 (W2.grad = grad_output.T @ hidden)
    dim3 block_size(32, 32);
    dim3 grid_size((HIDDEN_SIZE + block_size.x - 1) / block_size.x, (OUTPUT_SIZE + block_size.y - 1) / block_size.y);
    matmul_at_b_kernel<<<grid_size, block_size>>>(d_hidden, d_grad_output, nn->grad_weights2, batch_size, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Update gradients for bias2
    update_gradients_kernel<<<grid_size, block_size>>>(nn->grad_weights2, nn->grad_bias2, d_grad_output, d_hidden, batch_size, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Compute dX2 (gradient of loss w.r.t. input of second layer)
    float *d_dX2;
    CUDA_CHECK(hipMalloc(&d_dX2, batch_size * HIDDEN_SIZE * sizeof(float)));
    grid_size.x = (HIDDEN_SIZE + block_size.x - 1) / block_size.x;
    grid_size.y = (batch_size + block_size.y - 1) / block_size.y;
    matmul_a_bt_kernel<<<grid_size, block_size>>>(d_grad_output, nn->weights2, d_dX2, batch_size, OUTPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Compute d_ReLU_out (element-wise multiplication with ReLU derivative)
    float *d_grad_hidden;
    CUDA_CHECK(hipMalloc(&d_grad_hidden, batch_size * HIDDEN_SIZE * sizeof(float)));
    drelu_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, d_grad_hidden, batch_size * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());


    multiply_gradients_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_dX2, d_grad_hidden, batch_size * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Update gradients for weights1 (W1.grad = d_ReLU_out.T @ input)
    grid_size.x = (INPUT_SIZE + block_size.x - 1) / block_size.x;
    grid_size.y = (HIDDEN_SIZE + block_size.y - 1) / block_size.y;
    matmul_at_b_kernel<<<grid_size, block_size>>>(d_input, d_dX2, nn->grad_weights1, batch_size, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Update gradients for bias1
    update_gradients_kernel<<<grid_size, block_size>>>(nn->grad_weights1, nn->grad_bias1, d_dX2, d_input, batch_size, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Free allocated memory
    CUDA_CHECK(hipFree(d_grad_output));
    CUDA_CHECK(hipFree(d_dX2));
    CUDA_CHECK(hipFree(d_grad_hidden));

    CUDA_CHECK(hipDeviceSynchronize());
}

// gradient descent step
__global__ void update_weights_kernel(float *weights, float *grad_weights, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        weights[idx] -= LEARNING_RATE * grad_weights[idx];
    }
}

void update_weights(NeuralNetwork *nn) {
    int block_size = 256;
    int grid_size;

    // Update weights1
    grid_size = (HIDDEN_SIZE * INPUT_SIZE + block_size - 1) / block_size;
    update_weights_kernel<<<grid_size, block_size>>>(nn->weights1, nn->grad_weights1, HIDDEN_SIZE * INPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Update weights2
    grid_size = (OUTPUT_SIZE * HIDDEN_SIZE + block_size - 1) / block_size;
    update_weights_kernel<<<grid_size, block_size>>>(nn->weights2, nn->grad_weights2, OUTPUT_SIZE * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Update bias1
    grid_size = (HIDDEN_SIZE + block_size - 1) / block_size;
    update_weights_kernel<<<grid_size, block_size>>>(nn->bias1, nn->grad_bias1, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Update bias2
    grid_size = (OUTPUT_SIZE + block_size - 1) / block_size;
    update_weights_kernel<<<grid_size, block_size>>>(nn->bias2, nn->grad_bias2, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());
}

// Modified train function to work with CUDA
void train(NeuralNetwork *nn, float *X_train, int *y_train) {
    float *d_X_train, *d_hidden, *d_output;
    int *d_y_train;

    CUDA_CHECK(hipMalloc(&d_X_train, TRAIN_SIZE * INPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_hidden, BATCH_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y_train, TRAIN_SIZE * sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_X_train, X_train, TRAIN_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y_train, y_train, TRAIN_SIZE * sizeof(int), hipMemcpyHostToDevice));


    int num_batches = TRAIN_SIZE / BATCH_SIZE;

    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        float total_loss = 0.0f;
        int correct = 0;
        
        // Zero out gradients at the beginning of each epoch
        zero_grad_kernel<<<(HIDDEN_SIZE * INPUT_SIZE + 256 - 1) / 256, 256>>>(nn->grad_weights1, HIDDEN_SIZE * INPUT_SIZE);
        zero_grad_kernel<<<(OUTPUT_SIZE * HIDDEN_SIZE + 256 - 1) / 256, 256>>>(nn->grad_weights2, OUTPUT_SIZE * HIDDEN_SIZE);
        zero_grad_kernel<<<(HIDDEN_SIZE + 256 - 1) / 256, 256>>>(nn->grad_bias1, HIDDEN_SIZE);
        zero_grad_kernel<<<(OUTPUT_SIZE + 256 - 1) / 256, 256>>>(nn->grad_bias2, OUTPUT_SIZE);
        CUDA_CHECK(hipDeviceSynchronize());

        for (int batch = 0; batch < num_batches; batch++) {
            int start_idx = batch * BATCH_SIZE;
            
            forward(nn, &d_X_train[start_idx * INPUT_SIZE], d_hidden, d_output, BATCH_SIZE);

            float *h_output = (float *)malloc(BATCH_SIZE * OUTPUT_SIZE * sizeof(float));
            CUDA_CHECK(hipMemcpy(h_output, d_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost));

            float loss = cross_entropy_loss(h_output, &y_train[start_idx], BATCH_SIZE);
            total_loss += loss;

            for (int i = 0; i < BATCH_SIZE; i++) {
                int predicted = 0;
                for (int j = 1; j < OUTPUT_SIZE; j++) {
                    if (h_output[i * OUTPUT_SIZE + j] > h_output[i * OUTPUT_SIZE + predicted]) {
                        predicted = j;
                    }
                }
                if (predicted == y_train[start_idx + i]) {
                    correct++;
                }
            }



            free(h_output);

            backward(nn, &d_X_train[start_idx * INPUT_SIZE], d_hidden, d_output, &d_y_train[start_idx], BATCH_SIZE);

            update_weights(nn);

            if ((batch + 1) % 100 == 0 || (epoch == 0 && batch == 0)) {
                printf("Epoch %d/%d, Iter %d/%d, Loss: %.4f, Accuracy: %.2f%%\n", 
                       epoch + 1, EPOCHS, batch + 1, num_batches, total_loss / (batch + 1), 
                       100.0f * correct / ((batch + 1) * BATCH_SIZE));
            }
        }
        
        printf("Epoch %d/%d completed, Loss: %.4f, Accuracy: %.2f%%\n", 
            epoch + 1, EPOCHS, total_loss / num_batches, 100.0f * correct / TRAIN_SIZE);
    }
    
    CUDA_CHECK(hipFree(d_X_train));
    CUDA_CHECK(hipFree(d_hidden));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_y_train));
}

// Modified initialize function to allocate memory for gradients
void initialize_neural_network(NeuralNetwork *nn) {
    CUDA_CHECK(hipMalloc(&nn->weights1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->weights2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->bias1, HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->bias2, OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->grad_weights1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->grad_weights2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->grad_bias1, HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->grad_bias2, OUTPUT_SIZE * sizeof(float)));

    // Allocate temporary host memory
    float *h_weights1 = (float *)malloc(HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    float *h_weights2 = (float *)malloc(OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    float *h_bias1 = (float *)malloc(HIDDEN_SIZE * sizeof(float));
    float *h_bias2 = (float *)malloc(OUTPUT_SIZE * sizeof(float));

    // Initialize weights and biases on the host
    initialize_weights(h_weights1, HIDDEN_SIZE * INPUT_SIZE);
    initialize_weights(h_weights2, OUTPUT_SIZE * HIDDEN_SIZE);
    initialize_bias(h_bias1, HIDDEN_SIZE);
    initialize_bias(h_bias2, OUTPUT_SIZE);

    // Copy initialized values to device
    CUDA_CHECK(hipMemcpy(nn->weights1, h_weights1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(nn->weights2, h_weights2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(nn->bias1, h_bias1, HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(nn->bias2, h_bias2, OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Free temporary host memory
    free(h_weights1);
    free(h_weights2);
    free(h_bias1);
    free(h_bias2);
}

int main() {
    srand(time(NULL));

    NeuralNetwork nn;
    initialize_neural_network(&nn);

    float *X_train = (float *)malloc(TRAIN_SIZE * INPUT_SIZE * sizeof(float));
    int *y_train = (int *)malloc(TRAIN_SIZE * sizeof(int));
    float *X_test = (float *)malloc(TEST_SIZE * INPUT_SIZE * sizeof(float));
    int *y_test = (int *)malloc(TEST_SIZE * sizeof(int));

    load_data("../../mnist_data/X_train.bin", X_train, TRAIN_SIZE * INPUT_SIZE);
    load_labels("../../mnist_data/y_train.bin", y_train, TRAIN_SIZE);
    load_data("../../mnist_data/X_test.bin", X_test, TEST_SIZE * INPUT_SIZE);
    load_labels("../../mnist_data/y_test.bin", y_test, TEST_SIZE);


    // print first image in the terminal
    for (int i = 0; i < 28; i++) {
        for (int j = 0; j < 28; j++) {
            if (X_train[0 * INPUT_SIZE + i * 28 + j] > 0.0f) {
                printf("X");
            } else {
                printf(" ");
            }
        }
        printf("\n");
    }

    printf("First 10 training labels: ");
    for (int i = 0; i < 10; i++) {
        printf("%d ", y_train[i]);
    }
    printf("\n");
    
    train(&nn, X_train, y_train);

    CUDA_CHECK(hipFree(nn.weights1));
    CUDA_CHECK(hipFree(nn.weights2));
    CUDA_CHECK(hipFree(nn.bias1));
    CUDA_CHECK(hipFree(nn.bias2));
    CUDA_CHECK(hipFree(nn.grad_weights1));
    CUDA_CHECK(hipFree(nn.grad_weights2));
    CUDA_CHECK(hipFree(nn.grad_bias1));
    CUDA_CHECK(hipFree(nn.grad_bias2));
    free(X_train);
    free(y_train);
    free(X_test);
    free(y_test);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}
