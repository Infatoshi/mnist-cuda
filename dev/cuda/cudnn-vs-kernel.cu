#include "hip/hip_runtime.h"
#include "dataloader/dataloader.cuh"
#include <iostream>
#include <hipDNN.h>
#include <chrono>
#include <cmath>

using namespace std;

// CUDA error checking macro
#define CUDA_CHECK(error) \
    if (error != hipSuccess) { \
        cout << "CUDA error: " << hipGetErrorString(error) << " at " << __FILE__ << ":" << __LINE__ << endl; \
        exit(-1); \
    }

#define CUDNN_CHECK(status) \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        cout << "cuDNN error: " << hipdnnGetErrorString(status) << " at " << __FILE__ << ":" << __LINE__ << endl; \
        exit(-1); \
    }

// Custom CUDA kernel for "valid" 2D convolution -> output_size = input_size - kernel_size + 1
__global__ void custom_conv2d_kernel(float* input, float* output, float* kernel, int B, int C, int H, int W, int K, int KH, int KW) {
    int b = blockIdx.x;
    int k = blockIdx.y;
    int h = threadIdx.x;
    int w = threadIdx.y;

    float sum = 0.0f;
    for (int c = 0; c < C; ++c) {
        for (int kh = 0; kh < KH; ++kh) {
            for (int kw = 0; kw < KW; ++kw) {
                int ih = h + kh;
                int iw = w + kw;
                if (ih < H && iw < W) {
                    sum += input[b * C * H * W + c * H * W + ih * W + iw] * kernel[k * C * KH * KW + c * KH * KW + (KH - 1 - kh) * KW + (KW - 1 - kw)];
                }
            }
        }
    }
    output[b * K * (H-KH+1) * (W-KW+1) + k * (H-KH+1) * (W-KW+1) + h * (W-KW+1) + w] = sum;
}

// Function to compare outputs
bool compare_outputs(const float* output1, const float* output2, int size, float tolerance = 1e-2) {
    for (int i = 0; i < size; ++i) {
        if (fabs(output1[i] - output2[i]) > tolerance) {
            return false;
        }
    }
    return true;
}

void print_tensor_shape(const string& name, int n, int c, int h, int w) {
    cout << name << " shape: (" << n << ", " << c << ", " << h << ", " << w << ")" << endl;
}

void print_first_element(const string& name, const float* data, int c, int h, int w) {
    cout << name << " first element:" << endl;
    for (int i = 0; i < c; ++i) {
        cout << "- Channel " << i << ":" << endl;
        for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k) {
                cout << data[i * h * w + j * w + k] << " ";
            }
            cout << endl;
        }
    }
}

int main() {
    const int batch_size = 128;
    const int num_epochs = 5;
    const int data_size = 28 * 28; // MNIST image dimensions

    vector<float> train_data;
    vector<int> train_labels;

    // Load and normalize data
    read_mnist_image_file("../../data/MNIST/raw/train-images-idx3-ubyte", train_data);
    read_mnist_label_file("../../data/MNIST/raw/train-labels-idx1-ubyte", train_labels);
    normalize_data(train_data);

    // Allocate memory on GPU
    float* d_data;
    int* d_labels;
    allocate_gpu_memory(d_data, d_labels, train_data, train_labels);

    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    // Create tensor descriptors
    hipdnnTensorDescriptor_t input_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnTensorDescriptor_t output_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));

    // Set tensor descriptor
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, 28, 28));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 32, 1, 5, 5));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Find output dimensions
    int n, c, h, w;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc, &n, &c, &h, &w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

    // Print the output shape
    print_tensor_shape("cuDNN Convolution Output", n, c, h, w);
    
    // Allocate memory for the filter and output on GPU
    float* d_filter;
    float* d_output_cudnn;
    float* d_output_custom;

    CUDA_CHECK(hipMalloc(&d_filter, 32 * 1 * 5 * 5 * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output_cudnn, n * c * h * w * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output_custom, n * c * h * w * sizeof(float)));

    // Host memory to compare outputs
    float* h_output_cudnn = new float[n * c * h * w];
    float* h_output_custom = new float[n * c * h * w];

    // Random initialization of filters for demonstration purposes
    vector<float> h_filter(32 * 1 * 5 * 5);
    srand(42);
    for (float& el : h_filter) { el = static_cast<float>(rand()) / RAND_MAX; }
    CUDA_CHECK(hipMemcpy(d_filter, h_filter.data(), 32 * 1 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice));

    int iters_per_epoch = 60000 / batch_size;
    cout << "Iters per epoch: " << iters_per_epoch << endl;

    for (int epoch = 0; epoch < num_epochs; ++epoch) {
        for (int i = 0; i < iters_per_epoch; ++i) {
            float* batch_data;
            int* batch_labels;

            // Retrieve batch
            get_batch(d_data, d_labels, batch_data, batch_labels, batch_size, i, data_size);

            // Timing cuDNN Convolution
            auto start_cudnn = std::chrono::high_resolution_clock::now();
            // Perform cuDNN convolution
            const float alpha = 1.0f, beta = 0.0f;
            CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha, input_desc, batch_data, filter_desc, d_filter, conv_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, &beta, output_desc, d_output_cudnn));
            auto end_cudnn = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> time_cudnn = end_cudnn - start_cudnn;

            // Timing custom CUDA Convolution
            auto start_custom = std::chrono::high_resolution_clock::now();
            dim3 blockDim(h, w);
            dim3 gridDim(n, c);
            custom_conv2d_kernel<<<gridDim, blockDim>>>(batch_data, d_output_custom, d_filter, batch_size, 1, 28, 28, 32, 5, 5);
            CUDA_CHECK(hipDeviceSynchronize());
            auto end_custom = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> time_custom = end_custom - start_custom;

            // Copy results back to host
            CUDA_CHECK(hipMemcpy(h_output_cudnn, d_output_cudnn, n * c * h * w * sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(h_output_custom, d_output_custom, n * c * h * w * sizeof(float), hipMemcpyDeviceToHost));

            // Compare the results
            bool outputs_match = compare_outputs(h_output_cudnn, h_output_custom, n * c * h * w);

            // Print the output shape
            // print_tensor_shape("Custom CUDA Convolution Output", n, c, h, w);

            // Print the first element of the batch
            if (i == 0 && epoch == 0) {  // Print only for the first batch of the first epoch for brevity
                print_first_element("cuDNN Convolution First Element", h_output_cudnn, c, h, w);
                print_first_element("Custom CUDA Convolution First Element", h_output_custom, c, h, w);
            }

            if (i % 100 == 99) {
                cout << "Epoch: " << epoch + 1 << ", Iter: " << i + 1 << endl;
                cout << "cuDNN Conv2D time: " << time_cudnn.count() << " seconds" << endl;
                cout << "Custom CUDA Conv2D time: " << time_custom.count() << " seconds" << endl;
                cout << "Outputs match: " << (outputs_match ? "Yes" : "No") << endl;
            }
        }
    }

    cout << "Finished Training" << endl;

    // Free GPU memory
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_labels));
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_output_cudnn));
    CUDA_CHECK(hipFree(d_output_custom));

    // Free host memory
    delete[] h_output_cudnn;
    delete[] h_output_custom;

    // Destroy cuDNN descriptors and handle
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));

    return 0;
}