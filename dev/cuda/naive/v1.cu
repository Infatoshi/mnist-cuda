#include <iostream>
#include <cudnn.h>
#include <hip/hip_runtime.h>
#include <cmath> // for fabs

using namespace std;

// CUDA error checking macro
#define CUDA_CHECK(error) \
    if (error != hipSuccess) { \
        cout << "CUDA error: " << hipGetErrorString(error) << " at " << __FILE__ << ":" << __LINE__ << endl; \
        exit(-1); \
    }

#define CUDNN_CHECK(status) \
    if (status != CUDNN_STATUS_SUCCESS) { \
        cout << "cuDNN error: " << cudnnGetErrorString(status) << " at " << __FILE__ << ":" << __LINE__ << endl; \
        exit(-1); \
    }


